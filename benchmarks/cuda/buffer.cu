#include <hip/hip_runtime.h>
#include <nvm_types.h>
#include <nvm_dma.h>
#include <nvm_util.h>
#include <nvm_error.h>
#include <memory>
#include <stdexcept>
#include <string>
#include <new>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include "buffer.h"

using error = std::runtime_error;
using std::string;



static void getDeviceMemory(int device, void*& bufferPtr, void*& devicePtr, size_t size)
{
    bufferPtr = nullptr;
    devicePtr = nullptr;

    hipError_t err = hipSetDevice(device);
    if (err != hipSuccess)
    {
        throw error(string("Failed to set CUDA device: ") + hipGetErrorString(err));
    }

    err = hipMalloc(&bufferPtr, size);
    if (err != hipSuccess)
    {
        throw error(string("Failed to allocate device memory: ") + hipGetErrorString(err));
    }

    err = hipMemset(bufferPtr, 0, size);
    if (err != hipSuccess)
    {
        hipFree(bufferPtr);
        throw error(string("Failed to clear device memory: ") + hipGetErrorString(err));
    }

    hipPointerAttribute_t attrs;
    err = hipPointerGetAttributes(&attrs, bufferPtr);
    if (err != hipSuccess)
    {
        hipFree(bufferPtr);
        throw error(string("Failed to get pointer attributes: ") + hipGetErrorString(err));
    }

    devicePtr = attrs.devicePointer;
}



static void getDeviceMemory(int device, void*& bufferPtr, size_t size)
{
    void* notUsed = nullptr;
    getDeviceMemory(device, bufferPtr, notUsed, size);
}



DmaPtr createDma(const nvm_ctrl_t* ctrl, size_t size)
{
    nvm_dma_t* dma = nullptr;
    void* buffer = nullptr;

    hipError_t err = hipHostAlloc(&buffer, size, hipHostMallocDefault);
    if (err != hipSuccess)
    {
        throw error(string("Failed to allocate host memory: ") + hipGetErrorString(err));
    }

    int status = nvm_dma_map_host(&dma, ctrl, buffer, size);
    if (!nvm_ok(status))
    {
        hipHostFree(buffer);
        throw error(string("Failed to map host memory: ") + nvm_strerror(status));
    }

    return DmaPtr(dma, [buffer](nvm_dma_t* dma) {
        nvm_dma_unmap(dma);
        hipHostFree(buffer);
    });
}



DmaPtr createDma(const nvm_ctrl_t* ctrl, size_t size, int cudaDevice)
{
    if (cudaDevice < 0)
    {
        return createDma(ctrl, size);
    }

    nvm_dma_t* dma = nullptr;
    void* bufferPtr = nullptr;
    void* devicePtr = nullptr;

    getDeviceMemory(cudaDevice, bufferPtr, devicePtr, size);

    int status = nvm_dma_map_device(&dma, ctrl, (void *)NVM_PAGE_ALIGN((uintptr_t)devicePtr, 1UL << 16), NVM_ADDR_MASK(size, 1UL << 16));
    if (!nvm_ok(status))
    {
        hipFree(bufferPtr);
        throw error(string("Failed to map device memory: ") + nvm_strerror(status));
    }

    //dma->vaddr = bufferPtr;

    return DmaPtr(dma, [bufferPtr](nvm_dma_t* dma) {
        nvm_dma_unmap(dma);
        hipFree(bufferPtr);
    });
}



BufferPtr createBuffer(size_t size)
{
    void* buffer = nullptr;

    hipError_t err = hipHostAlloc(&buffer, size, hipHostMallocDefault);
    if (err != hipSuccess)
    {
        throw error(string("Failed to allocate host memory: ") + hipGetErrorString(err));
    }

    return BufferPtr(buffer, [](void* ptr) { hipHostFree(ptr); });
}



BufferPtr createBuffer(size_t size, int cudaDevice)
{
    if (cudaDevice < 0)
    {
        return createBuffer(size);
    }

    void* bufferPtr = nullptr;

    getDeviceMemory(cudaDevice, bufferPtr, size);

    return BufferPtr(bufferPtr, [](void* ptr) { hipFree(ptr); });
}



#ifdef __DIS_CLUSTER__
DmaPtr createDma(const nvm_ctrl_t* ctrl, size_t size, uint32_t adapter, uint32_t id)
{
    nvm_dma_t* dma = nullptr;

    int status = nvm_dis_dma_create(&dma, ctrl, adapter, id, size);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create local segment: ") + nvm_strerror(status));
    }

    return DmaPtr(dma, nvm_dma_unmap);
}
#else
DmaPtr createDma(const nvm_ctrl_t* ctrl, size_t size, uint32_t, uint32_t)
{
    return createDma(ctrl, size);
}
#endif


#ifdef __DIS_CLUSTER__
DmaPtr createDma(const nvm_ctrl_t* ctrl, size_t size, int cudaDevice, uint32_t adapter, uint32_t id)
{
    if (cudaDevice < 0)
    {
        return createDma(ctrl, size, adapter, id);
    }

    nvm_dma_t* dma = nullptr;
    void* bufferPtr = nullptr;
    void* devicePtr = nullptr;

    getDeviceMemory(cudaDevice, bufferPtr, devicePtr, size);

    int status = nvm_dis_dma_map_device(&dma, ctrl, adapter, id, devicePtr, size);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to create local segment: ") + nvm_strerror(status));
    }

    dma->vaddr = devicePtr;

    return DmaPtr(dma, nvm_dma_unmap);
}
#else
DmaPtr createDma(const nvm_ctrl_t* ctrl, size_t size, int cudaDevice, uint32_t, uint32_t)
{
    return createDma(ctrl, size, cudaDevice);
}
#endif


#ifdef __DIS_CLUSTER__
DmaPtr createRemoteDma(const nvm_ctrl_t* ctrl, size_t size, uint32_t adapter, uint32_t segno)
{
    nvm_dma_t* dma = nullptr;

    int status = nvm_dis_dma_connect(&dma, ctrl, adapter, segno, size, true);
    if (!nvm_ok(status))
    {
        throw error(string("Failed to connect to segment: ") + nvm_strerror(status));
    }

    return DmaPtr(dma, nvm_dma_unmap);
}
#endif

